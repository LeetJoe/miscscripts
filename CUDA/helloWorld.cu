
#include <hip/hip_runtime.h>
#include <cstdio>

// kernel definition
__global__ void HelloWorld() {
    printf("Hello, World!\n");
}

int main() {
    const auto nBlock = 1;
    const auto nThread = 1;
    HelloWorld<<< nBlock, nThread >>> ();
    return 0;
}