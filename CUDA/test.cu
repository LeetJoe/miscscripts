/* * @file_name HelloWorld.cu  后缀名称.cu */
#include <stdio.h>
#include <hip/hip_runtime.h>
//头文件

//核函数声明，前面的关键字__global__

__global__ void kernel( void ) {
}

int main1( void ) {
    //核函数的调用，注意<<<1,1>>>，第一个1，代表线程格里只有一个线程块；第二个1，代表一个线程块里只有一个线程。
    kernel<<<1,1>>>();
    printf( "Hello, World!\n" );
    return 0;
}

__global__ void add( int a, int b, int *c ) {
    *c = a + b;
}


int main( void ) {
    int c;
    int *dev_c;
    //cudaMalloc()
    hipMalloc( (void**)&dev_c, sizeof(int) );
    //核函数执行
    add<<<1,1>>>( 2, 7, dev_c );
    //cudaMemcpy()
    hipMemcpy( &c, dev_c, sizeof(int),hipMemcpyDeviceToHost ) ;
    printf( "2 + 7 = %d\n", c );
    //cudaFree()
    hipFree( dev_c );

    return 0;
}